#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <time.h>

extern "C" {
    #include "libs/bitmap.h"
}

#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__);  }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
       if (code != hipSuccess)
       {
                 fprintf(stderr,"GPUassert: %s %s %s %d\n", hipGetErrorName(code), hipGetErrorString(code), file, line);
                       if (abort) exit(code);
                          
       }
}

#define GRID_SIZE 16
#define BLOCK_SIZE 128
#define TOTAL_THREAD_COUNT GRID_SIZE * BLOCK_SIZE

// Convolutional Filter Examples, each with dimension 3,
// gaussian filter with dimension 5

int sobelYFilter[] = {-1, -2, -1,
                       0,  0,  0,
                       1,  2,  1};

int sobelXFilter[] = {-1, -0, 1,
                      -2,  0, 2,
                      -1,  0, 1};

int laplacian1Filter[] = { -1,  -4,  -1,
                           -4,  20,  -4,
                           -1,  -4,  -1};

int laplacian2Filter[] = { 0,  1,  0,
                           1, -4,  1,
                           0,  1,  0};

int laplacian3Filter[] = { -1,  -1,  -1,
                           -1,   8,  -1,
                           -1,  -1,  -1};

int gaussianFilter[] = { 1,  4,  6,  4, 1,
                         4, 16, 24, 16, 4,
                         6, 24, 36, 24, 6,
                         4, 16, 24, 16, 4,
                         1,  4,  6,  4, 1 };

const char* filterNames[]       = { "SobelY",     "SobelX",     "Laplacian 1",    "Laplacian 2",    "Laplacian 3",    "Gaussian"     };
int* const filters[]            = { sobelYFilter, sobelXFilter, laplacian1Filter, laplacian2Filter, laplacian3Filter, gaussianFilter };
unsigned int const filterDims[] = { 3,            3,            3,                3,                3,                5              };
float const filterFactors[]     = { 1.0,          1.0,          1.0,              1.0,              1.0,              1.0 / 256.0    };

int const maxFilterIndex = sizeof(filterDims) / sizeof(unsigned int);

void cleanup(char** input, char** output) {
    if (*input)
        free(*input);
    if (*output)
        free(*output);
}

void graceful_exit(char** input, char** output) {
    cleanup(input, output);
    exit(0);
}

void error_exit(char** input, char** output) {
    cleanup(input, output);
    exit(1);
}

// Helper function to swap bmpImageChannel pointers

void swapImageRawdata(pixel **one, pixel **two) {
  pixel *helper = *two;
  *two = *one;
  *one = helper;
}

void swapImage(bmpImage **one, bmpImage **two) {
  bmpImage *helper = *two;
  *two = *one;
  *one = helper;
}

// Apply convolutional filter on image data
void applyFilter(pixel *out, pixel *in, unsigned int width, unsigned int height, int *filter, unsigned int filterDim, float filterFactor) {
  unsigned int const filterCenter = (filterDim / 2);
  for (unsigned int y = 0; y < height; y++) {
    for (unsigned int x = 0; x < width; x++) {
      int ar = 0, ag = 0, ab = 0;
      for (unsigned int ky = 0; ky < filterDim; ky++) {
        int nky = filterDim - 1 - ky;
        for (unsigned int kx = 0; kx < filterDim; kx++) {
          int nkx = filterDim - 1 - kx;

          int yy = y + (ky - filterCenter);
          int xx = x + (kx - filterCenter);
          if (xx >= 0 && xx < (int) width && yy >=0 && yy < (int) height) {
            ar += in[yy*width + xx].r * filter[nky * filterDim + nkx];
            ag += in[yy*width + xx].g * filter[nky * filterDim + nkx];
            ab += in[yy*width + xx].b * filter[nky * filterDim + nkx];
          }
        }
      }

      ar *= filterFactor;
      ag *= filterFactor;
      ab *= filterFactor;
      
      ar = (ar < 0) ? 0 : ar;
      ag = (ag < 0) ? 0 : ag;
      ab = (ab < 0) ? 0 : ab;

      out[y*width +x].r = (ar > 255) ? 255 : ar;
      out[y*width +x].g = (ag > 255) ? 255 : ag;
      out[y*width +x].b = (ab > 255) ? 255 : ab;
    }
  }
}

// Apply convolutional filter on image data
__global__ void applyFilterDevice(pixel *out, pixel *in, unsigned int width, unsigned int height, int *filter, unsigned int filterDim, float filterFactor) {
  unsigned int const filterCenter = (filterDim / 2);
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x < width && y < height) {
      int ar = 0, ag = 0, ab = 0;
      for (unsigned int ky = 0; ky < filterDim; ky++) {
        int nky = filterDim - 1 - ky;
        for (unsigned int kx = 0; kx < filterDim; kx++) {
          int nkx = filterDim - 1 - kx;

          int yy = y + (ky - filterCenter);
          int xx = x + (kx - filterCenter);
          if (xx >= 0 && xx < (int) width && yy >=0 && yy < (int) height) {
            ar += in[yy*width + xx].r * filter[nky * filterDim + nkx];
            ag += in[yy*width + xx].g * filter[nky * filterDim + nkx];
            ab += in[yy*width + xx].b * filter[nky * filterDim + nkx];
          }
        }
      }

      ar *= filterFactor;
      ag *= filterFactor;
      ab *= filterFactor;
      
      ar = (ar < 0) ? 0 : ar;
      ag = (ag < 0) ? 0 : ag;
      ab = (ab < 0) ? 0 : ab;

      out[y*width +x].r = (ar > 255) ? 255 : ar;
      out[y*width +x].g = (ag > 255) ? 255 : ag;
      out[y*width +x].b = (ab > 255) ? 255 : ab;
  }
}

void help(char const *exec, char const opt, char const *optarg) {
    FILE *out = stdout;
    if (opt != 0) {
        out = stderr;
        if (optarg) {
            fprintf(out, "Invalid parameter - %c %s\n", opt, optarg);
        } else {
            fprintf(out, "Invalid parameter - %c\n", opt);
        }
    }
    fprintf(out, "%s [options] <input-bmp> <output-bmp>\n", exec);
    fprintf(out, "\n");
    fprintf(out, "Options:\n");
    fprintf(out, "  -k, --filter     <filter>        filter index (0<=x<=%u) (2)\n", maxFilterIndex -1);
    fprintf(out, "  -i, --iterations <iterations>    number of iterations (1)\n");

    fprintf(out, "\n");
    fprintf(out, "Example: %s before.bmp after.bmp -i 10000\n", exec);
}


int main(int argc, char **argv) {
  /*
    Parameter parsing, don't change this!
   */
  unsigned int iterations = 1;
  char *output = NULL;
  char *input = NULL;
  unsigned int filterIndex = 2;

  static struct option const long_options[] =  {
      {"help",       no_argument,       0, 'h'},
      {"filter",     required_argument, 0, 'k'},
      {"iterations", required_argument, 0, 'i'},
      {0, 0, 0, 0}
  };

  static char const * short_options = "hk:i:";
  {
    char *endptr;
    int c;
    int parse;
    int option_index = 0;
    while ((c = getopt_long(argc, argv, short_options, long_options, &option_index)) != -1) {
      switch (c) {
      case 'h':
        help(argv[0],0, NULL);
        graceful_exit(&input,&output);
      case 'k':
        parse = strtol(optarg, &endptr, 10);
        if (endptr == optarg || parse < 0 || parse >= maxFilterIndex) {
          help(argv[0], c, optarg);
          error_exit(&input,&output);
        }
        filterIndex = (unsigned int) parse;
        break;
      case 'i':
        iterations = strtol(optarg, &endptr, 10);
        if (endptr == optarg) {
          help(argv[0], c, optarg);
          error_exit(&input,&output);
        }
        break;
      default:
        abort();
      }
    }
  }

  if (argc <= (optind+1)) {
    help(argv[0],' ',"Not enough arugments");
    error_exit(&input,&output);
  }

  unsigned int arglen = strlen(argv[optind]);
  input = (char*)calloc(arglen + 1, sizeof(char));
  strncpy(input, argv[optind], arglen);
  optind++;

  arglen = strlen(argv[optind]);
  output = (char*)calloc(arglen + 1, sizeof(char));
  strncpy(output, argv[optind], arglen);
  optind++;

  /*
    End of Parameter parsing!
   */


  /*
    Create the BMP image and load it from disk.
   */
  bmpImage *image = newBmpImage(0,0);
  if (image == NULL) {
    fprintf(stderr, "Could not allocate new image!\n");
    error_exit(&input,&output);
  }

  if (loadBmpImage(image, input) != 0) {
    fprintf(stderr, "Could not load bmp image '%s'!\n", input);
    freeBmpImage(image);
    error_exit(&input,&output);
  }

  printf("Apply filter '%s' on image with %u x %u pixels for %u iterations\n", filterNames[filterIndex], image->width, image->height, iterations);


  // TODO: implement time measurement from here

  // Here we do the actual computation!
  // image->data is a 2-dimensional array of pixel which is accessed row first ([y][x])
  // image->rawdata is a 1-dimensional array of pixel containing the same data as image->data
  // each pixel is a struct of 3 unsigned char for the red, blue and green colour channel
  bmpImage *processImage = newBmpImage(image->width, image->height);

  // TODO: Cuda malloc and memcpy the rawdata from the images, from host side to device side
  pixel *devicePixelsIn;
  pixel *devicePixelsOut;
  int *filter;
  hipMalloc((void**)&devicePixelsIn, image->width * image->height * sizeof(pixel));
  hipMalloc((void**)&devicePixelsOut, image->width * image->height * sizeof(pixel));
  hipMalloc((void**)&filter, filterDims[filterIndex]*filterDims[filterIndex]*sizeof(int));
  hipMemcpy(devicePixelsIn, image->rawdata, image->width * image->height * sizeof(pixel), hipMemcpyHostToDevice);
  hipMemcpy(devicePixelsOut, processImage->rawdata, image->width * image->height * sizeof(pixel), hipMemcpyHostToDevice);
  hipMemcpy(filter, filters[filterIndex], filterDims[filterIndex]*filterDims[filterIndex]*sizeof(int), hipMemcpyHostToDevice);

  // TODO: Define the gridSize and blockSize, e.g. using dim3 (see Section 2.2. in CUDA Programming Guide)
  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks((image->width + threadsPerBlock.x - 1) / threadsPerBlock.x, (image->height + threadsPerBlock.y - 1) / threadsPerBlock.y);

  // TODO: Intialize and start CUDA timer
  clock_t t1 = clock();

  for (unsigned int i = 0; i < iterations; i ++) {
      // TODO: Implement kernel call instead of serial implementation
    applyFilterDevice<<<numBlocks,threadsPerBlock>>>(
        devicePixelsOut,
		    devicePixelsIn,
		    image->width,
		    image->height,
		    filter,
		    filterDims[filterIndex],
		    filterFactors[filterIndex]
    );
    //swapImage(&processImage, &image);
    swapImageRawdata(&devicePixelsOut, &devicePixelsIn);
  }

  hipError_t error = hipPeekAtLastError();
  if (error) {
      fprintf(stderr, "1: A CUDA error has occurred: %s\n", hipGetErrorString(error));
  }

  // TODO: Stop CUDA timer
  hipDeviceSynchronize();
  clock_t t2 = clock();

  // TODO: Copy back rawdata from images
  hipMemcpy(image->rawdata, devicePixelsIn, image->width * image->height * sizeof(pixel), hipMemcpyDeviceToHost);

  
  error = hipPeekAtLastError();
  if (error) {
      fprintf(stderr, "2: A CUDA error has occurred: %s\n", hipGetErrorString(error));
  }

  // TODO: Calculate and print elapsed time
  float spentTime = (t2 - t1)/CLOCKS_PER_SEC;
  printf("Time spent: %.3f seconds\n", spentTime);

  freeBmpImage(processImage);
  //Write the image back to disk
  if (saveBmpImage(image, output) != 0) {
    fprintf(stderr, "Could not save output to '%s'!\n", output);
    freeBmpImage(image);
    error_exit(&input,&output);
  };

  graceful_exit(&input,&output);
};
